#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
	
template <typename T>
std::string typeStr()
{
	if constexpr (std::is_same_v<T,float>)
		return "float";
	if constexpr (std::is_same_v<T,double>)
		return "double";
	if constexpr (std::is_same_v<T,int>)
		return "int";
}

template<typename T, int VEC>
struct VecType;

template<> struct VecType<float,4> { using type = float4; };
template<> struct VecType<int,4>   { using type = int4; };
template<> struct VecType<double,2>{ using type = double2; };

template <typename T>
__global__ void initArray(int N, T* v, T val)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = gridDim.x * blockDim.x;
	for (int i=tid; i<N; i+=stride)
		v[i] = val;
}

template <typename T>
__global__ void initArrayUnroll4(int N, T* v, T val)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = gridDim.x * blockDim.x;
#pragma unroll 4
	for (int i=tid; i<N; i+=stride)
		v[i] = val;
}

template <typename T>
__global__ void initArrayManualUnroll4(int N, T* v, T val)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x * gridDim.x;
	int i = tid;
	int full_stride = 4*stride;
	int limit = (N/full_stride)*full_stride;
	for (; i<limit; i+=full_stride)
	{
		v[i] = val;
		v[i+stride] = val;
		v[i+2*stride] = val;
		v[i+3*stride] = val;
	}
	//last elements
	for (; i <N; i+=stride)
		v[i] = val;
}

template <typename T>
__global__ void initArrayVec4(int N, T* v, T val)
{
	using VecT = typename VecType<T,4>::type;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	// reinterpret pointer as vector type
	VecT* v_v = reinterpret_cast<VecT*>(v);

	VecT val4;
	#pragma unroll
	for (int i=0; i<4; i++) {
		reinterpret_cast<T*>(&val4)[i] = val;
	}

	// number of full vectors
	size_t N_vec = N / 4;
	for (int i=tid; i<N_vec; i+=stride)
		v_v[i] = val4;

	//last elements
	for (int i = N_vec*4 + tid; i<N; i+=stride)
		v[i] = val;

}
#define XSTR(x) STR(x)
#define STR(x) #x

#ifndef KERNEL_NAME
#define KERNEL_NAME initArray
#endif

#ifndef DATATYPE
#define DATATYPE float
#endif

#ifndef NITERS
#define NITERS 1000
#endif

#ifndef NWARMUPS
#define NWARMUPS 20
#endif

int main(int argc, char **argv)
{
	using dtype = DATATYPE;

	int N = 1<<20; //array size
	dtype value = static_cast<dtype>(2.03f);

	int nthreads = 128;
	int nblocks = 128;

	if (argc > 1)
		N = atoi(argv[1]);
	if (argc > 2)
		nthreads = atoi(argv[2]);
	if (argc > 3)
		nblocks = atoi(argv[3]);

	size_t size = sizeof(dtype)*N;

	float current_time = 0.0f, total_time = 0.0f;

	std::cout << "ARRAY SIZE   " << N << "\n";
	std::cout << "DATATYPE     " << typeStr<dtype>() << "\n";
	std::cout << "MEMORY SIZE  " << size/1024./1024. << " MB\n";
	std::cout << "KERNEL       " << XSTR(KERNEL_NAME) << "\n";
	std::cout << "#THREADS     " << nthreads << "\n";
	std::cout << "#BLOCKS      " << nblocks << "\n";
	if constexpr (XSTR(KERNEL_NAME) == "initArray")
		std::cout << "#ELTS/THR    " << (double)N/double(nthreads*nblocks) << "\n";
	else
		std::cout << "#ELTS/THR    " << (double)N/double(nthreads)*nblocks*4 << "\n";
	dtype *data;
	hipMalloc(&data, size);

	for (int i=0; i<NWARMUPS; i++)
		KERNEL_NAME<<<nblocks, nthreads>>>(N, data, value);

	hipEvent_t start,end;

	hipEventCreate(&start);
	hipEventCreate(&end);
	
	std::cout << "ITERATIONS   " << NITERS << "\n";
	for (int i=0; i<NITERS; i++)
	{
		hipEventRecord(start);
		KERNEL_NAME<<<nblocks, nthreads>>>(N, data, value);
		hipEventRecord(end);
		hipEventSynchronize(end);
		hipEventElapsedTime(&current_time, start, end);
		total_time += current_time;
	}
	std::cout << "AVG. TIME    " << total_time*1000./(float)NITERS << " µs\n";

	hipFree(data);
	hipEventDestroy(start);
	hipEventDestroy(end);
}
